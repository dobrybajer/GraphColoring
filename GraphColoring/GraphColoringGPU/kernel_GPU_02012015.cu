#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <Windows.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#pragma region Headers

hipError_t runCuda(int*, int*, int, int);
hipError_t FindChromaticNumberMain(int*, int*, int*, int, int);
hipError_t initIndepSet(int, int*, int, int*, int, int, int*, int*, int*, int, int, int);
int* BuildingIndependentSetsGPU(int N, int* Vertices, int* Offest, int verticesLength);

#pragma endregion Headers

#pragma region Algorithm

	__host__ __device__ unsigned long Pow(int a, int n)
	{
		unsigned long result = 1;

		while (n)
		{
			if (n & 1)
				result *= a;
			
			n >>= 1;
			a *= a;
		}

		return result;
	}

	__host__ __device__ int sgnPow(int n)
	{
		return (n & 1) == 0 ? 1 : -1;
	}

	// Sprawdzi�, dlaczego to dzia�a
	__host__ __device__ int BitCount(int u)
	{
		int uCount = u - ((u >> 1) & 033333333333) - ((u >> 2) & 011111111111);
		return ((uCount + (uCount >> 3)) & 030707070707) % 63;
	}

	// Sprawdzi�, czy mo�na lepiej
	__host__ __device__ int Combination_n_of_k(int n, int k)
	{
		if (k > n) return 0;

		int r = 1;
		for (int d = 1; d <= k; ++d)
		{
			r *= n--;
			r /= d;
		}
		return r;
	} 

	int* BuildingIndependentSets(int N, int* Vertices, int* Offset)
	{
		int n = N;
		int* vertices = Vertices;
		int* offset = Offset;

		int* independentSets;
		int* actualVertices;
		int actualVerticesRowCount;
		int actualVerticesColCount;

		// Inicjalizacja macierzy o rozmiarze 2^N (warto�ci pocz�tkowe 0)
		independentSets = new int[1 << n] ();

		// Krok 1 algorytmu: przypisanie warto�ci 1 (ilo�� niezale�nych zbior�w) dla podzbior�w 1-elementowych, oraz dodanie ich do aktualnie przetwarzanych element�w (1 poziom tworzenia wszystkich podzbior�w)
		actualVertices = new int[n];

		actualVerticesRowCount = n;//oldRow	
		actualVerticesColCount = 1;//oldCol
		
		for (int i = 0; i < n; ++i)
		{
			independentSets[1 << i] = 1;
			actualVertices[i] = i;
		}

		// G��wna funkcja tworz�ca tablic� liczno�ci zbior�w niezale�nych dla wszystkich podzbior�w zbioru N-elementowego.
		// Zaczynamy od 1, bo krok pierwszy wykonany wy�ej.
		for (int el = 1; el < n; el++)
		{
			int col = el + 1;
			int row = Combination_n_of_k(n, col);
			int* newVertices = new int[row*col];//zmiana na tablice jedno wymiarow� 
			int l = 0;

			for (int i = 0; i < actualVerticesRowCount; ++i)
			{
				int lastIndex = 0;
				// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
				for (int index = 0; index < actualVerticesColCount; ++index)
					lastIndex += (1 << actualVertices[i*actualVerticesColCount + index]);
				for (int j = actualVertices[i*actualVerticesColCount + actualVerticesColCount - 1] + 1; j < n; ++j)
				{
					int lastIndex2 = lastIndex;
					// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
					for (int ns = offset[j - 1]; ns < offset[j]; ++ns)
					{
						for (int q = 0; q < actualVerticesColCount; ++q)
						{
							if (actualVertices[i * actualVerticesColCount + q] == vertices[ns])
							{
								lastIndex2 -= (1 << vertices[ns]);
								break;
							}
						}		
					}
					int nextIndex = lastIndex + (1 << j);
					// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
					independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;
					for (int k = 0; k < el; ++k)
						newVertices[l*col + k] = actualVertices[i * actualVerticesColCount + k];
					newVertices[l * col + el] = j;

					l++;
				}
			}		
			delete[] actualVertices;

			actualVertices = newVertices;

			actualVerticesRowCount = row;
			actualVerticesColCount = col;

		}
		return independentSets;
	}

	int* BuildingIndependentSetsGPU(int N, int* Vertices, int* Offest, int verticesLength)
{
		int n = N;
		int* vertices = Vertices;
		int* offset = Offest;

		int* independentSets;
		int* actualVertices;//zmiana na tablice jedno wymiarow� 
		int actualVerticesRowCount;
		int actualVerticesColCount;

		// Inicjalizacja macierzy o rozmiarze 2^N (warto�ci pocz�tkowe 0)
		independentSets = new int[1 << n] ();

		// Krok 1 algorytmu: przypisanie warto�ci 1 (ilo�� niezale�nych zbior�w) dla podzbior�w 1-elementowych, oraz dodanie ich do aktualnie przetwarzanych element�w (1 poziom tworzenia wszystkich podzbior�w)
		//CreateActualVertices(n, 1);
		actualVertices = new int[n];

		actualVerticesRowCount = n;//oldRow	
		actualVerticesColCount = 1;//oldCol
		
		for (int i = 0; i < n; ++i)
		{
			independentSets[1 << i] = 1;
			actualVertices[i] = i;
		}

		// G��wna funkcja tworz�ca tablic� liczno�ci zbior�w niezale�nych dla wszystkich podzbior�w zbioru N-elementowego.
		// Zaczynamy od 1, bo krok pierwszy wykonany wy�ej.
		for (int el = 1; el < n; el++)
		{	
			cout<<"row "<<actualVerticesRowCount<<endl;
			int col = el + 1;
			int row = Combination_n_of_k(n, col);
			int* newVertices = new int[row*col];//zmiana na tablice jedno wymiarow� 
		
			int l = 0;
			int roz=1<<N;
			
			initIndepSet(N,Vertices,verticesLength,Offest,actualVerticesRowCount,actualVerticesColCount,
				actualVertices,newVertices,independentSets,row,col,el);
		
			delete[] actualVertices;

			actualVertices = newVertices;
		
			actualVerticesRowCount = row;
			actualVerticesColCount = col;
			cout<<"nr "<<el<<endl;
    
		}
		return independentSets;
}

	__global__ void FindChromaticNumber(int N, int* independentSets, int* wynik)
	{
		int n = N;
		int index = threadIdx.x;

		unsigned long s = 0;
		int PowerNumber = 1 << n;

		for (int i = 0; i < PowerNumber; ++i) s += (sgnPow(BitCount(i)) * Pow(independentSets[i], index + 1));
			
		wynik[index] = s > 0 ? index : s; // KAMIL: punkt krytyczny, czy dobrze jest liczone "s"? dla unsigned long long liczy �le...
	}

	__global__ void IndependentSetGPU(int N ,int* Vertices,int* Offset ,int actualVerticesRowCount,
		int actualVerticesColCount,int* actualVertices,int* newVertices,int* independentSets,int col, int el )
	{
		int n = N;

		int i=threadIdx.x;
		int l = 0;

			int lastIndex = 0;
			// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
			for (int index = 0; index < actualVerticesColCount; ++index)
				lastIndex += (1 << actualVertices[i*actualVerticesColCount + index]);
			for (int j = actualVertices[i*actualVerticesColCount + actualVerticesColCount - 1] + 1; j < n; ++j)
			{
				int lastIndex2 = lastIndex;
				// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
				for (int ns = Offset[j - 1]; ns < Offset[j]; ++ns)
				{
					for (int q = 0; q < actualVerticesColCount; ++q)
					{
						if (actualVertices[i * actualVerticesColCount + q] == Vertices[ns])
						{
							lastIndex2 -= (1 << Vertices[ns]);
							break;
						}
					}		
				}
				int nextIndex = lastIndex + (1 << j);
				// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
				independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;
				for (int k = 0; k < el; ++k)
					newVertices[l*col + k] = actualVertices[i * actualVerticesColCount + k];
				newVertices[l * col + el] = j;
				
				l++;
			}
	
	}

#pragma endregion Algorithm

#pragma region Structure

	struct Graph
{
	int* vertices;
	int* neighbors;
	int n;
	int allVerticesCount;
};

	Graph ReadGraph(string path)
{
	fstream plik;
	plik.open(path, ios::in | ios::out);

	if (plik.good())
	{
		string line;
		getline(plik, line);

		int size = stoi(line);
		int i = 0, k = 0;
		int* nNeighborsCount = new int[size];
		vector<string> el;

		while (!plik.eof())
		{
			getline(plik, line);

			stringstream ss(line);
			string item;

			while (getline(ss, item, ','))
				el.push_back(item);

			nNeighborsCount[i] = el.size();

			k = el.size();
			i++;
		}
		plik.close();

		int* nVertices = new int[k];

		for (int i = 0; i < k; i++)
			nVertices[i] = stoi(el[i]);

		Graph g = { nVertices, nNeighborsCount, size, k };

		return g;
	}
	else throw new logic_error("Podczas otwierania pliku wyst�pi� b��d");
}

#pragma endregion Structure

#pragma region Time Measuring

double get_wall_time()
{
	LARGE_INTEGER time, freq;
	if (!QueryPerformanceFrequency(&freq)) { return 0; }
	if (!QueryPerformanceCounter(&time)) { return 0; }
	return (double)time.QuadPart / freq.QuadPart;
}

double get_cpu_time()
{
	FILETIME a, b, c, d;
	if (GetProcessTimes(GetCurrentProcess(), &a, &b, &c, &d) != 0)
		return (double)(d.dwLowDateTime | ((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
	else
		return 0;
}

#pragma endregion Time Measuring

int main()
{
	Graph graph = ReadGraph("../../TestFiles/GraphExample12.txt");

	//int roz = 1 << graph.n;

	//int* independentSet = BuildingIndependentSets(graph.n, graph.vertices, graph.neighbors);
	//int* independentSet = BuildingIndependentSetsGPU(graph.n, graph.vertices, graph.neighbors, graph.allVerticesCount);

	int* tabWyn = new int[graph.n];

	double wall0 = get_wall_time();
	double cpu0 = get_cpu_time();

	//hipError_t cudaStatus = runCuda(tabWyn, independentSet, graph.n, roz);
	hipError_t cudaStatus = FindChromaticNumberMain(tabWyn, graph.vertices, graph.neighbors, graph.n, graph.allVerticesCount);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int wynik = 0;

	for(int i = 0; i < graph.n; i++)
	{
		if(tabWyn[i]!=-1 && tabWyn[i]!=0)
		{
			wynik = tabWyn[i] + 1;
			break;
		}
	}

	double wall1 = get_wall_time();
	double cpu1 = get_cpu_time();

	cout << "Wall Time = " << wall1 - wall0 << " seconds" << endl;
	cout << "CPU Time  = " << cpu1 - cpu0 << " seconds" << endl;

	cout << "Potrzeba " << wynik << " kolorow." << endl;
	
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

#pragma region CudaFunctions

hipError_t runCuda(int *wynik, int *independentSet, int sizeWynik, int sizeIndep)
{
    int *dev_independentSet = 0;
    int *dev_wynik = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));
    
    // Allocate GPU buffers for three vectors (two input, one output)    .
    gpuErrchk(hipMalloc((void**)&dev_wynik, sizeWynik * sizeof(int)));

    gpuErrchk(hipMalloc((void**)&dev_independentSet, sizeIndep * sizeof(int)));
    
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_independentSet, independentSet, sizeIndep * sizeof(int), hipMemcpyHostToDevice));
    
    // Launch a kernel on the GPU with one thread for each element.
	FindChromaticNumber<<<1,sizeWynik>>>(sizeWynik, dev_independentSet, dev_wynik);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    gpuErrchk(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(wynik, dev_wynik, sizeWynik * sizeof(int), hipMemcpyDeviceToHost));

    return cudaStatus;
}

hipError_t initIndepSet(int N, int* Vertices, int verticeslength, int* Offset, int actualVerticesRowCount,
		int actualVerticesColCount, int* actualVertices, int* newVertices, int* independentSets, int row, int col, int el)
{
	int *dev_Vertices = 0;
    int *dev_Offset = 0;
	int *dev_independentSets=0;
	int *dev_actualVertices=0;
	int *dev_newVertices=0;	
    hipError_t cudaStatus=hipSuccess;
	int roz= 1<<N;
	//cout<<roz<<endl;
	gpuErrchk(hipSetDevice(0));

	cout<<"length"<<verticeslength<<endl;
	gpuErrchk(hipMalloc((void**)&dev_Vertices, verticeslength * sizeof(int)));
   
	gpuErrchk(hipMalloc((void**)&dev_Offset, N * sizeof(int)));
    
    gpuErrchk(hipMalloc((void**)&dev_independentSets, roz * sizeof(int)));
 
	gpuErrchk(hipMalloc((void**)&dev_actualVertices, (actualVerticesColCount*actualVerticesRowCount) * sizeof(int)));
 
	gpuErrchk(hipMalloc((void**)&dev_newVertices, (row*col) * sizeof(int)));

	gpuErrchk(hipMemcpy(dev_independentSets, independentSets, roz * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_Vertices, Vertices, verticeslength * sizeof(int), hipMemcpyHostToDevice));
 
	gpuErrchk(hipMemcpy(dev_Offset, Offset, N * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_actualVertices, actualVertices, actualVerticesColCount*actualVerticesRowCount * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_newVertices, newVertices, row*col * sizeof(int), hipMemcpyHostToDevice));
  
	
	IndependentSetGPU<<<1,actualVerticesRowCount>>>( N , dev_Vertices, dev_Offset , actualVerticesRowCount,
		 actualVerticesColCount,dev_actualVertices, dev_newVertices, dev_independentSets, col, el);
	
	hipDeviceSynchronize();

	  gpuErrchk(hipGetLastError());
      
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    gpuErrchk(hipDeviceSynchronize());
   
    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(independentSets, dev_independentSets, roz * sizeof(int), hipMemcpyDeviceToHost));
    	
	gpuErrchk(hipMemcpy(actualVertices, dev_actualVertices, actualVerticesColCount*actualVerticesRowCount * sizeof(int), hipMemcpyDeviceToHost));

	gpuErrchk(hipMemcpy(newVertices, dev_newVertices, row*col * sizeof(int), hipMemcpyDeviceToHost));
   
	/*hipFree(dev_actualVertices);
	hipFree(dev_independentSets);
    hipFree(dev_newVertices);
	hipFree(dev_Offset);
	hipFree(dev_Vertices);*/

    return cudaStatus;
}

#pragma endregion CudaFunctions

#pragma region CudaFunctions - version 2
// Final
__global__ void Init(int* independentSet, int* actualVertices, int verticesCount)
{
	int PowerNumber = 1 << verticesCount;

	for(int i = 0; i < PowerNumber; ++i)
		independentSet[i] = 0;

	for (int i = 0; i < verticesCount; ++i)
	{
		independentSet[1 << i] = 1;
		actualVertices[i] = i;
	}
}

// Final
__global__ void CreateActualVertices(int* actualVertices, int* newVertices, int size)
{
	for(int i = 0; i < size; ++i)
		actualVertices[i] = newVertices[i];
}

// Mo�liwe zmiany, je�li b�dzie lepszy pomys�
__global__ void PrepareToNewVertices(int* actualVertices, int* l_set, int n, int actualVerticesRowCount, int actualVerticesColCount)
{
	int last_el = 0;
	int last_index = 0;
	l_set[0] = 0;

	for(int i = 1; i < actualVerticesRowCount; ++i)
	{
		int previousPossibleCombination = n - actualVertices[last_index * actualVerticesColCount + actualVerticesColCount - 1] - 1;
		int actualPossibleCombination = n - actualVertices[i * actualVerticesColCount + actualVerticesColCount - 1] - 1;
	
		if(actualPossibleCombination <= 0)
			l_set[i] = -1;
		else
		{
			l_set[i] = last_el + previousPossibleCombination;
			last_el = l_set[i];
			last_index = i;
		}
	}
}

// Konieczne zmiany po wprowadzeniu blok�w, a mo�liwe �e i siatek (grid)
__global__ void BuildIndependentSetGPU(int* l_set, int n, int* vertices, int* offset, int actCol, int newCol, int* actualVertices, int* newVertices, int* independentSets)
{
	int i = threadIdx.x;
	int l = l_set[i];

	if (l==-1) return;

	int lastIndex = 0;

	for (int index = 0; index < actCol; ++index)
		lastIndex += (1 << actualVertices[i * actCol + index]);

	for (int j = actualVertices[i * actCol + actCol - 1] + 1; j < n; ++j)
	{
		int lastIndex2 = lastIndex;

		for (int ns = offset[j - 1]; ns < offset[j]; ++ns)
		{
			for (int q = 0; q < actCol; ++q)
			{
				if (actualVertices[i * actCol + q] == vertices[ns])
				{
					lastIndex2 -= (1 << vertices[ns]);
					break;
				}
			}		
		}

		int nextIndex = lastIndex + (1 << j);

		independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;

		for (int k = 0; k < newCol - 1; ++k)
			newVertices[l * newCol + k] = actualVertices[i * actCol + k];

		newVertices[l * newCol + newCol - 1] = j;
				
		l++;
	}
}

// Do sprawdzenia szczeg�lnie kwestia alokowanej i zwalnianej pami�ci
hipError_t FindChromaticNumberMain(int* wynik, int* vertices, int* offset, int verticesCount, int allVerticesCount)
{
    int* dev_vertices = 0;
	int* dev_offset = 0;
    int* dev_wynik = 0;

	int* dev_independentSet = 0;
	int* dev_actualVertices = 0;
	int* dev_newVertices = 0;
	int* dev_l_set = 0;
	int actualVerticesRowCount = verticesCount;
	int actualVerticesColCount = 1;
	int PowerNumber = 1 << verticesCount;

    hipError_t cudaStatus = hipSuccess;

    gpuErrchk(hipSetDevice(0));

	gpuErrchk(hipMalloc((void**)&dev_vertices, allVerticesCount * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_offset, verticesCount * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_wynik, verticesCount * sizeof(int)));

	gpuErrchk(hipMalloc((void**)&dev_independentSet, PowerNumber * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_actualVertices, verticesCount * sizeof(int)));

    gpuErrchk(hipMemcpy(dev_vertices, vertices, allVerticesCount * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_offset, offset, verticesCount * sizeof(int), hipMemcpyHostToDevice));
    
	Init<<<1,1>>> (dev_independentSet, dev_actualVertices, verticesCount); // czy warto odpali� na wi�kszej ilo�ci w�tk�w? (wpisywanie du�ej ilo�ci zer)

	for (int el = 1; el < verticesCount; el++) // przy tej konstrukcji alg nie damy rady odpali� tej p�tli r�wnolegle
	{	
		int col = el + 1;
		int row = Combination_n_of_k(verticesCount, col);

		gpuErrchk(hipMalloc((void**)&dev_newVertices, (row * col) * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&dev_l_set, actualVerticesRowCount * sizeof(int)));
		
		PrepareToNewVertices<<<1,1>>> (dev_actualVertices, dev_l_set, verticesCount, actualVerticesRowCount, actualVerticesColCount); // przy tej konstrukcji funkcji nie damy rady odpali� tego na wielu w�tkach

		BuildIndependentSetGPU<<<1,actualVerticesRowCount>>> (dev_l_set, verticesCount, dev_vertices, dev_offset, actualVerticesColCount, col, dev_actualVertices, dev_newVertices, dev_independentSet); // Koniecznie trzeba odpala� tak�e u�ywaj�c blok�w. Max w�tk�w per blok to np. 1024, a s� sytuacje gdzie podawane jest ponad 180k (dla n=20)	

		hipFree(dev_actualVertices); // czy aby na pewno dobrze jest pamiec zwalniana? nie marnujemy zasobow karty?
		gpuErrchk(hipMalloc((void**)&dev_actualVertices, (row * col) * sizeof(int))); // czy ponowne mallocowanie jest ok je�li wcze�niej u�yto hipFree?

		CreateActualVertices<<<1,1>>> (dev_actualVertices, dev_newVertices, row * col);

		actualVerticesRowCount = row;
		actualVerticesColCount = col;
	}
	
	FindChromaticNumber<<<1,verticesCount>>> (verticesCount, dev_independentSet, dev_wynik); // Mo�liwe odpalenie blok�w, czyli zrobienie Reduce dla pewnych kawa�k�w ca�ej sumy. Ponadto komunikacja- przerwanie oblicze� natychmiast, gdy jaki� w�tek/blok da� pozytywn� odpowied�
	
    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(wynik, dev_wynik, verticesCount * sizeof(int), hipMemcpyDeviceToHost));

    return cudaStatus;
}

#pragma endregion CudaFunctions - version 2