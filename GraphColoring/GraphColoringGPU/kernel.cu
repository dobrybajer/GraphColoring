#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "Graph.h"

#include <stdio.h>
#include <iostream>

using namespace version_cpu;
using namespace std;

hipError_t runCuda(int *c, int *a,   int sizec, int sizea);

#pragma region Algorithm

	 __host__ __device__ unsigned long Pow(int a, int n)
	{
		unsigned long result = 1;

		while (n)
		{
			if (n & 1)
				result *= a;
			
			n >>= 1;
			a *= a;
		}

		return result;
	}

	// Final
	 __host__ __device__ int sgnPow(int n)
	{
		return (n & 1) == 0 ? 1 : -1;
	}

	// Sprawdzi�, dlaczego to dzia�a
	 __host__ __device__ int BitCount(int u)
	{
		int uCount = u - ((u >> 1) & 033333333333) - ((u >> 2) & 011111111111);
		return ((uCount + (uCount >> 3)) & 030707070707) % 63;
	}

	// Sprawdzi�, czy mo�na lepiej
	 __host__ __device__ int Combination_n_of_k(int n, int k)
	{
		if (k > n) return 0;

		int r = 1;
		for (int d = 1; d <= k; ++d)
		{
			r *= n--;
			r /= d;
		}
		return r;
	} 

	  int* BuildingIndependentSets(int N ,int* Vertices,int* Offest )
	{
		int n = N;
		int* vertices = Vertices;
		int* offset = Offest;

		int* independentSets;
		int** actualVertices;
		int actualVerticesRowCount;
		int actualVerticesColCount;

		// Inicjalizacja macierzy o rozmiarze 2^N (warto�ci pocz�tkowe 0)
		independentSets = new int[1 << n] ();

		// Krok 1 algorytmu: przypisanie warto�ci 1 (ilo�� niezale�nych zbior�w) dla podzbior�w 1-elementowych, oraz dodanie ich do aktualnie przetwarzanych element�w (1 poziom tworzenia wszystkich podzbior�w)
		//CreateActualVertices(n, 1);
		actualVertices = new int*[n];

		for (int i = 0; i < n; ++i)
			actualVertices[i] = new int[1] ();

		actualVerticesRowCount = n;
		actualVerticesColCount = 1;
		
		for (int i = 0; i < n; ++i)
		{
			independentSets[1 << i] = 1;//gubienie dla samych zer
			actualVertices[i][0] = i;
		}

		// G��wna funkcja tworz�ca tablic� liczno�ci zbior�w niezale�nych dla wszystkich podzbior�w zbioru N-elementowego.
		// Zaczynamy od 1, bo krok pierwszy wykonany wy�ej.
		for (int el = 1; el < n; el++)
		{
			
			int col = el + 1;
			int row = Combination_n_of_k(n, col);
			
			//int** newVertices = CreateNewVertices(row, col);
			int** newVertices = new int*[row];
			for (int i = 0; i < row; ++i)
				newVertices[i] = new int[col]();

			int l = 0;

			for (int i = 0; i < actualVerticesRowCount; ++i)
			{
				int lastIndex = 0;

				// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
				for (int index = 0; index < actualVerticesColCount; ++index)
					lastIndex += (1 << actualVertices[i][index]);

				for (int j = actualVertices[i][actualVerticesColCount - 1] + 1; j < n; ++j)
				{
					int lastIndex2 = lastIndex;

					// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
					for (int ns = offset[j - 1]; ns < offset[j]; ++ns)
					{
						for (int q = 0; q < actualVerticesColCount; ++q)
						{
							if (actualVertices[i][q] == vertices[ns])
							{
								lastIndex2 -= (1 << vertices[ns]);
								break;
							}
						}		
					}

					int nextIndex = lastIndex + (1 << j);

					// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
					independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;

					for (int k = 0; k < el; ++k)
						newVertices[l][k] = actualVertices[i][k];

					newVertices[l][el] = j;

					l++;
				}
			}
			//UpdateActualVertices(newVertices, row, col);
			for (int i = 0; i < actualVerticesRowCount; ++i)
			{
				delete[] actualVertices[i];
			}
			delete[] actualVertices;

			actualVertices = newVertices;

			actualVerticesRowCount = row;
			actualVerticesColCount = col;
		}
		return independentSets;
	}

	__global__ void FindChromaticNumber(int N ,int* independentSets,int *wynik)
	{
		int n = N;
		int index= threadIdx.x;

			unsigned long s = 0;
			int PowerNumber = Pow(2, n);
			// Czy mo�na omin�� u�ycie funkcji BitCount ?
			for (int i = 0; i < PowerNumber; ++i) s += (sgnPow(BitCount(i)) * Pow(independentSets[i], index));
			
			if (s > 0)
				wynik[index]=index;
			else
				wynik[index]=-1;
		
	}
#pragma endregion Algorithm

int main()
{
	Graph graph = Graph();
	graph= Graph::ReadGraph("test.txt");

	int roz=0;

	roz=1<<graph.GetVerticesCount();

	int* independentSet = BuildingIndependentSets(graph.GetVerticesCount(),graph.GetVertices(),graph.GetNeighborsCount());

	int* tabWyn=new int[graph.GetVerticesCount()];

	hipError_t cudaStatus = runCuda(tabWyn, independentSet, graph.GetVerticesCount(),roz);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int wynik =0;
	for(int i =0 ; i<graph.GetVerticesCount();i++)
		if(tabWyn[i]!=-1)
		{
			wynik = tabWyn[i];
			break;
		}

	cout<<"Potrzeba "<<wynik<<" kolorow"<<endl;
	

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t runCuda(int *wynik, int *independentSet,   int sizeWynik, int sizeIndep)
{
    int *dev_independentSet = 0;
    int *dev_wynik = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_wynik, sizeWynik * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_independentSet, sizeIndep * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_independentSet, independentSet, sizeIndep * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	FindChromaticNumber<<<1,sizeWynik>>>(sizeWynik,dev_independentSet,dev_wynik);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(wynik, dev_wynik, sizeWynik * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_wynik);
    hipFree(dev_independentSet);
    
    return cudaStatus;
}
