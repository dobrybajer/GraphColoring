#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#pragma region Headers

hipError_t runCuda(int*, int*, int, int);
hipError_t runCuda2(int*, int*, int*, int, int);
hipError_t initIndepSet(int, int*, int, int*, int, int, int*, int*, int*, int, int, int);
int* BuildingIndependentSetsGPU(int N, int* Vertices, int* Offest, int verticesLength);

#pragma endregion Headers

#pragma region Algorithm

	__host__ __device__ unsigned long Pow(int a, int n)
	{
		unsigned long result = 1;

		while (n)
		{
			if (n & 1)
				result *= a;
			
			n >>= 1;
			a *= a;
		}

		return result;
	}

	__host__ __device__ int sgnPow(int n)
	{
		return (n & 1) == 0 ? 1 : -1;
	}

	// Sprawdzi�, dlaczego to dzia�a
	__host__ __device__ int BitCount(int u)
	{
		int uCount = u - ((u >> 1) & 033333333333) - ((u >> 2) & 011111111111);
		return ((uCount + (uCount >> 3)) & 030707070707) % 63;
	}

	// Sprawdzi�, czy mo�na lepiej
	__host__ __device__ int Combination_n_of_k(int n, int k)
	{
		if (k > n) return 0;

		int r = 1;
		for (int d = 1; d <= k; ++d)
		{
			r *= n--;
			r /= d;
		}
		return r;
	} 

	int* BuildingIndependentSets(int N, int* Vertices, int* Offset)
	{
		int n = N;
		int* vertices = Vertices;
		int* offset = Offset;

		int* independentSets;
		int* actualVertices;
		int actualVerticesRowCount;
		int actualVerticesColCount;

		// Inicjalizacja macierzy o rozmiarze 2^N (warto�ci pocz�tkowe 0)
		independentSets = new int[1 << n] ();

		// Krok 1 algorytmu: przypisanie warto�ci 1 (ilo�� niezale�nych zbior�w) dla podzbior�w 1-elementowych, oraz dodanie ich do aktualnie przetwarzanych element�w (1 poziom tworzenia wszystkich podzbior�w)
		actualVertices = new int[n];

		actualVerticesRowCount = n;//oldRow	
		actualVerticesColCount = 1;//oldCol
		
		for (int i = 0; i < n; ++i)
		{
			independentSets[1 << i] = 1;
			actualVertices[i] = i;
		}

		// G��wna funkcja tworz�ca tablic� liczno�ci zbior�w niezale�nych dla wszystkich podzbior�w zbioru N-elementowego.
		// Zaczynamy od 1, bo krok pierwszy wykonany wy�ej.
		for (int el = 1; el < n; el++)
		{
			int col = el + 1;
			int row = Combination_n_of_k(n, col);
			int* newVertices = new int[row*col];//zmiana na tablice jedno wymiarow� 
			int l = 0;

			for (int i = 0; i < actualVerticesRowCount; ++i)
			{
				int lastIndex = 0;
				// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
				for (int index = 0; index < actualVerticesColCount; ++index)
					lastIndex += (1 << actualVertices[i*actualVerticesColCount + index]);
				for (int j = actualVertices[i*actualVerticesColCount + actualVerticesColCount - 1] + 1; j < n; ++j)
				{
					int lastIndex2 = lastIndex;
					// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
					for (int ns = offset[j - 1]; ns < offset[j]; ++ns)
					{
						for (int q = 0; q < actualVerticesColCount; ++q)
						{
							if (actualVertices[i * actualVerticesColCount + q] == vertices[ns])
							{
								lastIndex2 -= (1 << vertices[ns]);
								break;
							}
						}		
					}
					int nextIndex = lastIndex + (1 << j);
					// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
					independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;
					for (int k = 0; k < el; ++k)
						newVertices[l*col + k] = actualVertices[i * actualVerticesColCount + k];
					newVertices[l * col + el] = j;

					l++;
				}
			}		
			delete[] actualVertices;

			actualVertices = newVertices;

			actualVerticesRowCount = row;
			actualVerticesColCount = col;

		}
		return independentSets;
	}

	int* BuildingIndependentSetsGPU(int N, int* Vertices, int* Offest, int verticesLength)
{
		int n = N;
		int* vertices = Vertices;
		int* offset = Offest;

		int* independentSets;
		int* actualVertices;//zmiana na tablice jedno wymiarow� 
		int actualVerticesRowCount;
		int actualVerticesColCount;

		// Inicjalizacja macierzy o rozmiarze 2^N (warto�ci pocz�tkowe 0)
		independentSets = new int[1 << n] ();

		// Krok 1 algorytmu: przypisanie warto�ci 1 (ilo�� niezale�nych zbior�w) dla podzbior�w 1-elementowych, oraz dodanie ich do aktualnie przetwarzanych element�w (1 poziom tworzenia wszystkich podzbior�w)
		//CreateActualVertices(n, 1);
		actualVertices = new int[n];

		actualVerticesRowCount = n;//oldRow	
		actualVerticesColCount = 1;//oldCol
		
		for (int i = 0; i < n; ++i)
		{
			independentSets[1 << i] = 1;
			actualVertices[i] = i;
		}

		// G��wna funkcja tworz�ca tablic� liczno�ci zbior�w niezale�nych dla wszystkich podzbior�w zbioru N-elementowego.
		// Zaczynamy od 1, bo krok pierwszy wykonany wy�ej.
		for (int el = 1; el < n; el++)
		{	
			cout<<"row "<<actualVerticesRowCount<<endl;
			int col = el + 1;
			int row = Combination_n_of_k(n, col);
			int* newVertices = new int[row*col];//zmiana na tablice jedno wymiarow� 
		
			int l = 0;
			int roz=1<<N;
			
			initIndepSet(N,Vertices,verticesLength,Offest,actualVerticesRowCount,actualVerticesColCount,
				actualVertices,newVertices,independentSets,row,col,el);
		
			delete[] actualVertices;

			actualVertices = newVertices;
		
			actualVerticesRowCount = row;
			actualVerticesColCount = col;
			cout<<"nr "<<el<<endl;
    
		}
		return independentSets;
}

	__global__ void FindChromaticNumber(int N, int* independentSets, int* wynik)
	{
		int n = N;
		int index = threadIdx.x;

		unsigned long s = 0;
		int PowerNumber = 1 << n;

		for (int i = 0; i < PowerNumber; ++i) s += (sgnPow(BitCount(i)) * Pow(independentSets[i], index + 1));
			
		wynik[index] = s > 0 ? index : s; // KAMIL: punkt krytyczny, czy dobrze jest liczone "s"? dla unsigned long long liczy �le...
	}

	__global__ void IndependentSetGPU(int N ,int* Vertices,int* Offset ,int actualVerticesRowCount,
		int actualVerticesColCount,int* actualVertices,int* newVertices,int* independentSets,int col, int el )
	{
		int n = N;

		int i=threadIdx.x;
		int l = 0;

			int lastIndex = 0;
			// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
			for (int index = 0; index < actualVerticesColCount; ++index)
				lastIndex += (1 << actualVertices[i*actualVerticesColCount + index]);
			for (int j = actualVertices[i*actualVerticesColCount + actualVerticesColCount - 1] + 1; j < n; ++j)
			{
				int lastIndex2 = lastIndex;
				// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
				for (int ns = Offset[j - 1]; ns < Offset[j]; ++ns)
				{
					for (int q = 0; q < actualVerticesColCount; ++q)
					{
						if (actualVertices[i * actualVerticesColCount + q] == Vertices[ns])
						{
							lastIndex2 -= (1 << Vertices[ns]);
							break;
						}
					}		
				}
				int nextIndex = lastIndex + (1 << j);
				// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
				independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;
				for (int k = 0; k < el; ++k)
					newVertices[l*col + k] = actualVertices[i * actualVerticesColCount + k];
				newVertices[l * col + el] = j;
				
				l++;
			}
	
	}

#pragma endregion Algorithm

#pragma region Structure

	struct Graph
{
	int* vertices;
	int* neighbors;
	int n;
	int allVerticesCount;
};

	Graph ReadGraph(string path)
{
	fstream plik;
	plik.open(path, ios::in | ios::out);

	if (plik.good())
	{
		string line;
		getline(plik, line);

		int size = stoi(line);
		int i = 0, k = 0;
		int* nNeighborsCount = new int[size];
		vector<string> el;

		while (!plik.eof())
		{
			getline(plik, line);

			stringstream ss(line);
			string item;

			while (getline(ss, item, ','))
				el.push_back(item);

			nNeighborsCount[i] = el.size();

			k = el.size();
			i++;
		}
		plik.close();

		int* nVertices = new int[k];

		for (int i = 0; i < k; i++)
			nVertices[i] = stoi(el[i]);

		Graph g = { nVertices, nNeighborsCount, size, k };

		return g;
	}
	else throw new logic_error("Podczas otwierania pliku wyst�pi� b��d");
}

#pragma endregion Structure

int main()
{
	Graph graph = ReadGraph("../../TestFiles/GraphExampleMyciel3.txt");

	//int roz = 1 << graph.n;

	//int* independentSet = BuildingIndependentSets(graph.n, graph.vertices, graph.neighbors);
	//int* independentSet = BuildingIndependentSetsGPU(graph.n, graph.vertices, graph.neighbors, graph.allVerticesCount);

	int* tabWyn = new int[graph.n];

	//hipError_t cudaStatus = runCuda(tabWyn, independentSet, graph.n, roz);
	hipError_t cudaStatus = runCuda2(tabWyn, graph.vertices, graph.neighbors, graph.n, graph.allVerticesCount);

	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int wynik = 0;

	for(int i = 0; i < graph.n; i++)
	{
		if(tabWyn[i]!=-1 && tabWyn[i]!=0)
		{
			wynik = tabWyn[i] + 1;
			break;
		}
	}

	//for(int i=0;i<graph.n;i++)
	//	cout << " " << tabWyn[i];

	cout << endl << "Potrzeba " << wynik << " kolorow." << endl;
	
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

#pragma region CudaFunctions

hipError_t runCuda(int *wynik, int *independentSet, int sizeWynik, int sizeIndep)
{
    int *dev_independentSet = 0;
    int *dev_wynik = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));
    
    // Allocate GPU buffers for three vectors (two input, one output)    .
    gpuErrchk(hipMalloc((void**)&dev_wynik, sizeWynik * sizeof(int)));

    gpuErrchk(hipMalloc((void**)&dev_independentSet, sizeIndep * sizeof(int)));
    
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_independentSet, independentSet, sizeIndep * sizeof(int), hipMemcpyHostToDevice));
    
    // Launch a kernel on the GPU with one thread for each element.
	FindChromaticNumber<<<1,sizeWynik>>>(sizeWynik, dev_independentSet, dev_wynik);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    gpuErrchk(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(wynik, dev_wynik, sizeWynik * sizeof(int), hipMemcpyDeviceToHost));

    return cudaStatus;
}

hipError_t initIndepSet(int N, int* Vertices, int verticeslength, int* Offset, int actualVerticesRowCount,
		int actualVerticesColCount, int* actualVertices, int* newVertices, int* independentSets, int row, int col, int el)
{
	int *dev_Vertices = 0;
    int *dev_Offset = 0;
	int *dev_independentSets=0;
	int *dev_actualVertices=0;
	int *dev_newVertices=0;	
    hipError_t cudaStatus=hipSuccess;
	int roz= 1<<N;
	//cout<<roz<<endl;
	gpuErrchk(hipSetDevice(0));

	cout<<"length"<<verticeslength<<endl;
	gpuErrchk(hipMalloc((void**)&dev_Vertices, verticeslength * sizeof(int)));
   
	gpuErrchk(hipMalloc((void**)&dev_Offset, N * sizeof(int)));
    
    gpuErrchk(hipMalloc((void**)&dev_independentSets, roz * sizeof(int)));
 
	gpuErrchk(hipMalloc((void**)&dev_actualVertices, (actualVerticesColCount*actualVerticesRowCount) * sizeof(int)));
 
	gpuErrchk(hipMalloc((void**)&dev_newVertices, (row*col) * sizeof(int)));

	gpuErrchk(hipMemcpy(dev_independentSets, independentSets, roz * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_Vertices, Vertices, verticeslength * sizeof(int), hipMemcpyHostToDevice));
 
	gpuErrchk(hipMemcpy(dev_Offset, Offset, N * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_actualVertices, actualVertices, actualVerticesColCount*actualVerticesRowCount * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(dev_newVertices, newVertices, row*col * sizeof(int), hipMemcpyHostToDevice));
  
	
	IndependentSetGPU<<<1,actualVerticesRowCount>>>( N , dev_Vertices, dev_Offset , actualVerticesRowCount,
		 actualVerticesColCount,dev_actualVertices, dev_newVertices, dev_independentSets, col, el);
	
	hipDeviceSynchronize();

	  gpuErrchk(hipGetLastError());
      
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    gpuErrchk(hipDeviceSynchronize());
   
    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(independentSets, dev_independentSets, roz * sizeof(int), hipMemcpyDeviceToHost));
    	
	gpuErrchk(hipMemcpy(actualVertices, dev_actualVertices, actualVerticesColCount*actualVerticesRowCount * sizeof(int), hipMemcpyDeviceToHost));

	gpuErrchk(hipMemcpy(newVertices, dev_newVertices, row*col * sizeof(int), hipMemcpyDeviceToHost));
   
	/*hipFree(dev_actualVertices);
	hipFree(dev_independentSets);
    hipFree(dev_newVertices);
	hipFree(dev_Offset);
	hipFree(dev_Vertices);*/

    return cudaStatus;
}

#pragma endregion CudaFunctions

#pragma region CudaFunctions - version 2

__global__ void Init1(int* independentSet, int* actualVertices, int verticesCount)
{
	for (int i = 0; i < verticesCount; ++i)
	{
		independentSet[1 << i] = 1;
		actualVertices[i] = i;
	}
}

__global__ void Init2(int* actualVertices, int* newVertices, int size)
{
	for(int i = 0; i < size; ++i)
		actualVertices[i] = newVertices[i];
}

__global__ void Init3(int* actualVertices, int* l_set, int n, int actualVerticesRowCount, int actualVerticesColCount)
{
	int last_el = 0;
	int last_index = 0;
	l_set[0] = 0;
	for(int i = 1; i < actualVerticesRowCount; ++i)
	{
		int j = n - actualVertices[(last_index) * actualVerticesColCount + actualVerticesColCount - 1] - 1;
		int actual = n - actualVertices[i * actualVerticesColCount + actualVerticesColCount - 1] - 1;
	
		if(actual <= 0)
			l_set[i] = -1;
		else
		{
			l_set[i] = last_el + j;
			last_el = l_set[i];
			last_index = i;
		}
	}
}

__global__ void IndependentSetGPU2(int* l_set, int n, int* Vertices, int* Offset, int actualVerticesRowCount, int actualVerticesColCount, int* actualVertices, int* newVertices, int* independentSets, int col, int el)
{
	int i = threadIdx.x;
	int l = l_set[i];

	if (l==-1) return;

	int lastIndex = 0;
	// Sprawdzenie indeksu poporzedniego zbioru dla rozpatrywanego podzbioru
	for (int index = 0; index < actualVerticesColCount; ++index)
		lastIndex += (1 << actualVertices[i * actualVerticesColCount + index]);

	for (int j = actualVertices[i * actualVerticesColCount + actualVerticesColCount - 1] + 1; j < n; ++j)
	{
		int lastIndex2 = lastIndex;
		// Sprawdzenie indeksu poprzedniego zbioru dla rozpatrywanego podzbioru \ {i}
		for (int ns = Offset[j - 1]; ns < Offset[j]; ++ns)
		{
			for (int q = 0; q < actualVerticesColCount; ++q)
			{
				if (actualVertices[i * actualVerticesColCount + q] == Vertices[ns])
				{
					lastIndex2 -= (1 << Vertices[ns]);
					break;
				}
			}		
		}
		int nextIndex = lastIndex + (1 << j);
		// Liczba zbior�w niezale�nych w aktualnie przetwarzanym podzbiorze
		independentSets[nextIndex] = independentSets[lastIndex] + independentSets[lastIndex2] + 1;
		for (int k = 0; k < el; ++k)
			newVertices[l*col + k] = actualVertices[i * actualVerticesColCount + k];
		newVertices[l * col + el] = j;
				
		l++;
	}
}

hipError_t runCuda2(int* wynik, int* vertices, int* offset, int verticesCount, int allVerticesCount)
{
    int* dev_vertices = 0;
	int* dev_offset = 0;
    int* dev_wynik = 0;

	int* dev_independentSet = 0;
	int* dev_actualVertices = 0;
	int* dev_newVertices = 0;
	int* dev_l_set = 0;
	int actualVerticesRowCount = verticesCount;
	int actualVerticesColCount = 1;
	int PowerNumber = 1 << verticesCount;

    hipError_t cudaStatus = hipSuccess;

    gpuErrchk(hipSetDevice(0));

	gpuErrchk(hipMalloc((void**)&dev_vertices, allVerticesCount * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_offset, verticesCount * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_wynik, verticesCount * sizeof(int)));

	gpuErrchk(hipMalloc((void**)&dev_independentSet, PowerNumber * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_actualVertices, verticesCount * sizeof(int)));

    gpuErrchk(hipMemcpy(dev_vertices, vertices, allVerticesCount * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_offset, offset, verticesCount * sizeof(int), hipMemcpyHostToDevice));
    
	Init1<<<1,1>>> (dev_independentSet, dev_actualVertices, verticesCount);

	for (int el = 1; el < verticesCount; el++)
	{	
		/*int* a_set = new int[actualVerticesRowCount*actualVerticesColCount];
		gpuErrchk(hipMemcpy(a_set, dev_actualVertices, actualVerticesRowCount*actualVerticesColCount * sizeof(int), hipMemcpyDeviceToHost));
		cout << "a_set"<<endl;
		for(int r = 0; r < actualVerticesRowCount*actualVerticesColCount; ++r)
			cout << a_set[r] << " ";
		cout << endl;*/

		int col = el + 1;
		int row = Combination_n_of_k(verticesCount, col);

		gpuErrchk(hipMalloc((void**)&dev_newVertices, (row * col) * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&dev_l_set, actualVerticesRowCount * sizeof(int)));
		
		Init3<<<1,1>>> (dev_actualVertices, dev_l_set, verticesCount, actualVerticesRowCount, actualVerticesColCount);

		/*int* l_set = new int[actualVerticesRowCount];
		gpuErrchk(hipMemcpy(l_set, dev_l_set, actualVerticesRowCount * sizeof(int), hipMemcpyDeviceToHost));
		cout << "l_set"<<endl;
		for(int r = 0; r < actualVerticesRowCount; ++r)
			cout << l_set[r] << " ";
		cout << endl;*/

		IndependentSetGPU2<<<1,actualVerticesRowCount>>> (dev_l_set, verticesCount, dev_vertices, dev_offset, actualVerticesRowCount, actualVerticesColCount, dev_actualVertices, dev_newVertices, dev_independentSet, col, el);	

		//int* n_set = new int[row*col];
		//gpuErrchk(hipMemcpy(n_set, dev_newVertices, row*col * sizeof(int), hipMemcpyDeviceToHost));
		//cout << "n_set"<<endl;
		//for(int r = 0; r < row*col; ++r)
		//	cout << n_set[r] << " ";
		//cout << endl;

		hipFree(dev_actualVertices);
		gpuErrchk(hipMalloc((void**)&dev_actualVertices, (row * col) * sizeof(int)));
		Init2<<<1,1>>> (dev_actualVertices, dev_newVertices, row * col);

		actualVerticesRowCount = row;
		actualVerticesColCount = col;
		cout << "nr " <<el << endl;
	}
	
	FindChromaticNumber<<<1,verticesCount>>>(verticesCount, dev_independentSet, dev_wynik);

    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(wynik, dev_wynik, verticesCount * sizeof(int), hipMemcpyDeviceToHost));

    return cudaStatus;
}

#pragma endregion CudaFunctions - version 2